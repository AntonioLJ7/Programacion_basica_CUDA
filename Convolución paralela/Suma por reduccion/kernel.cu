﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define N 2048

// Realiza la suma secuencial de los valores del vector
float sumaSecuencial(float* vector)
{
	float fResultado = 0.0;

	for (int iPos = 0; iPos < N; iPos++)
		fResultado += vector[iPos];

	return fResultado;
}


__global__ void sumaParalela(float* vector)
{
	__shared__ float vectorCompartido[N];

	vectorCompartido[threadIdx.x] = vector[threadIdx.x];

	if (threadIdx.x + blockDim.x < N) {
		vectorCompartido[threadIdx.x + blockDim.x] = vector[threadIdx.x + blockDim.x];
		
		__syncthreads();
	}


	for (unsigned int iPos = N >> 1; iPos >= 1; iPos = iPos >> 1)
	{
		if (threadIdx.x < iPos)
			vectorCompartido[threadIdx.x] += vectorCompartido[threadIdx.x + iPos];
			
		__syncthreads();
	}

	if (threadIdx.x == 0)
		vector[0] = vectorCompartido[0];
}

int main(void)
{
	float host_v[N];
	float fResultadoParalelo, fResultadoSecuencial;
	float* dev_v;

	// Se llena de forma aleatoria el vector sobre el que se realiza la suma
	srand((unsigned)time(NULL));
	for (int i = 0; i < N; i++)
		host_v[i] = floorf(100 * (rand() / (float)RAND_MAX));

	// Pedir memoria en el Device para el vector a sumar (dev_v)
	/* COMPLETAR */

	hipMalloc((void**)&dev_v, N * sizeof(float));

	// Transferir el vector del Host al Device
	/* COMPLETAR */

	hipMemcpy(dev_v, host_v, N * sizeof(float), hipMemcpyHostToDevice);

	int threads = (N / 2) + N % 2;

	// Llamar al kernell CUDA
	/* COMPLETAR */

	sumaParalela <<<1, threads>>> (dev_v);

	// Copiar el resultado de la operación paralela del Device al Host
	/* COMPLETAR */

	hipMemcpy(&fResultadoParalelo, dev_v,sizeof(float), hipMemcpyDeviceToHost); // Solo necesitamos 1 float

	// Se comprueba que el resultado es correcto y se muestra un mensaje
	fResultadoSecuencial = sumaSecuencial(host_v);
	if (fResultadoParalelo == fResultadoSecuencial)
		printf("Operacion correcta\nDevice = %f\nHost   = %f\n", fResultadoParalelo, fResultadoSecuencial);
	else
		printf("Operacion INCORRECTA\nDevice = %f\nHost =   %f\n", fResultadoParalelo, fResultadoSecuencial);

	// Librerar la memoria solicitada en el Device
	/* COMPLETAR */

	hipFree(dev_v);

	return 0;
}