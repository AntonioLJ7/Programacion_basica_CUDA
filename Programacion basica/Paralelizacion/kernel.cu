#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>


#define N 10


__global__ void add(float* a, float* b, float* c)
{

    int tid =  threadIdx.x;

    c[tid] = a[tid] + b[tid];

}

int main()
{

    float host_a[N], host_b[N], host_c[N];
    float* dev_a, * dev_b, * dev_c;

    srand((unsigned)time(NULL));

    for (int i = 0; i < N; i++) {
        host_a[i] = floorf(1000*(rand()/(float)RAND_MAX));
        host_b[i] = floorf(1000*(rand()/(float)RAND_MAX));
    }

    //Rserva memoria en el Device
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));

    // Tranfiere una zona de memoria del Host al Device
    hipMemcpy(dev_a, host_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, N * sizeof(float), hipMemcpyHostToDevice);

    //Llamada al kernell (add) con parametros (dev_a, dev_b, dev_c)
    //El primer parametro (N) indica el numero de bloques de ejecucion que se crearan en el Device 
    //El segundo parametro (1) indica el numero de hebras de cada bloque (512)
    add << <1,N>> > (dev_a, dev_b, dev_c);

    //Transfiere una zona de memoria del Device al Host
    hipMemcpy(host_c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        printf("%f + %f = %f\n", host_a[i], host_b[i], host_c[i]);

    // Se libera la memoria reservada en el Device
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}